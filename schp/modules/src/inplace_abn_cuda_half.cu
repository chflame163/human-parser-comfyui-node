#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_fp16.h>

#include <vector>

#include "utils/checks.h"
#include "utils/cuda.cuh"
#include "inplace_abn.h"

#include <ATen/cuda/HIPContext.h>

// Operations for reduce
struct SumOpH {
  __device__ SumOpH(const half *t, int c, int s)
      : tensor(t), chn(c), sp(s) {}
  __device__ __forceinline__ float operator()(int batch, int plane, int n) {
    return __half2float(tensor[(batch * chn + plane) * sp + n]);
  }
  const half *tensor;
  const int chn;
  const int sp;
};

struct VarOpH {
  __device__ VarOpH(float m, const half *t, int c, int s)
      : mean(m), tensor(t), chn(c), sp(s) {}
  __device__ __forceinline__ float operator()(int batch, int plane, int n) {
    const auto t = __half2float(tensor[(batch * chn + plane) * sp + n]);
    return (t - mean) * (t - mean);
  }
  const float mean;
  const half *tensor;
  const int chn;
  const int sp;
};

struct GradOpH {
  __device__ GradOpH(float _weight, float _bias, const half *_z, const half *_dz, int c, int s)
      : weight(_weight), bias(_bias), z(_z), dz(_dz), chn(c), sp(s) {}
  __device__ __forceinline__ Pair<float> operator()(int batch, int plane, int n) {
    float _y = (__half2float(z[(batch * chn + plane) * sp + n]) - bias) / weight;
    float _dz = __half2float(dz[(batch * chn + plane) * sp + n]);
    return Pair<float>(_dz, _y * _dz);
  }
  const float weight;
  const float bias;
  const half *z;
  const half *dz;
  const int chn;
  const int sp;
};

// 添加类型转换辅助函数
template<typename T>
__device__ __forceinline__ half to_half(T x) {
    return __float2half(static_cast<float>(x));
}

template<typename T>
__device__ __forceinline__ float from_half(T x) {
    return __half2float(x);
}

/***********
 * mean_var
 ***********/

__global__ void mean_var_kernel_h(const c10::Half *x, float *mean, float *var, int num, int chn, int sp) {
  int plane = blockIdx.x;
  float norm = 1.f / static_cast<float>(num * sp);

  float _mean = reduce<float, SumOpH>(SumOpH(reinterpret_cast<const half*>(x), chn, sp), plane, num, sp) * norm;
  __syncthreads();
  float _var = reduce<float, VarOpH>(VarOpH(_mean, reinterpret_cast<const half*>(x), chn, sp), plane, num, sp) * norm;

  if (threadIdx.x == 0) {
    mean[plane] = _mean;
    var[plane] = _var;
  }
}

std::vector<at::Tensor> mean_var_cuda_h(at::Tensor x) {
  CHECK_CUDA_INPUT(x);

  // Extract dimensions
  int64_t num, chn, sp;
  get_dims(x, num, chn, sp);

  // Prepare output tensors
  auto mean = at::empty({chn},x.options().dtype(at::kFloat));
  auto var = at::empty({chn},x.options().dtype(at::kFloat));

  // Run kernel
  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  auto stream = at::cuda::getCurrentCUDAStream();
  mean_var_kernel_h<<<blocks, threads, 0, stream>>>(
      x.data_ptr<c10::Half>(),
      mean.data_ptr<float>(),
      var.data_ptr<float>(),
      num, chn, sp);

  return {mean, var};
}

/**********
 * forward
 **********/

__global__ void forward_kernel_h(c10::Half *x, const float *mean, const float *var, const float *weight, const float *bias,
                                 bool affine, float eps, int num, int chn, int sp) {
  int plane = blockIdx.x;

  const float _mean = mean[plane];
  const float _var = var[plane];
  const float _weight = affine ? abs(weight[plane]) + eps : 1.f;
  const float _bias = affine ? bias[plane] : 0.f;

  const float mul = rsqrt(_var + eps) * _weight;

  for (int batch = 0; batch < num; ++batch) {
    for (int n = threadIdx.x; n < sp; n += blockDim.x) {
      c10::Half *x_ptr = x + (batch * chn + plane) * sp + n;
      float _x = __half2float(*reinterpret_cast<half*>(x_ptr));
      float _y = (_x - _mean) * mul + _bias;

      *reinterpret_cast<half*>(x_ptr) = __float2half(_y);
    }
  }
}

at::Tensor forward_cuda_h(at::Tensor x, at::Tensor mean, at::Tensor var, at::Tensor weight, at::Tensor bias,
                        bool affine, float eps) {
  CHECK_CUDA_INPUT(x);
  CHECK_CUDA_INPUT(mean);
  CHECK_CUDA_INPUT(var);
  CHECK_CUDA_INPUT(weight);
  CHECK_CUDA_INPUT(bias);

  // Extract dimensions
  int64_t num, chn, sp;
  get_dims(x, num, chn, sp);

  // Run kernel
  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  auto stream = at::cuda::getCurrentCUDAStream();
  forward_kernel_h<<<blocks, threads, 0, stream>>>(
      x.data_ptr<c10::Half>(),
      mean.data_ptr<float>(),
      var.data_ptr<float>(),
      weight.data_ptr<float>(),
      bias.data_ptr<float>(),
      affine, eps, num, chn, sp);

  return x;
}

__global__ void edz_eydz_kernel_h(const c10::Half *z, const c10::Half *dz, const float *weight, const float *bias,
                                float *edz, float *eydz, bool affine, float eps, int num, int chn, int sp) {
  int plane = blockIdx.x;

  float _weight = affine ? abs(weight[plane]) + eps : 1.f;
  float _bias = affine ? bias[plane] : 0.f;

  Pair<float> res = reduce<Pair<float>, GradOpH>(GradOpH(_weight, _bias, 
    reinterpret_cast<const half*>(z), 
    reinterpret_cast<const half*>(dz), chn, sp), plane, num, sp);
  __syncthreads();

  if (threadIdx.x == 0) {
    edz[plane] = res.v1;
    eydz[plane] = res.v2;
  }
}

std::vector<at::Tensor> edz_eydz_cuda_h(at::Tensor z, at::Tensor dz, at::Tensor weight, at::Tensor bias,
                                      bool affine, float eps) {
  CHECK_CUDA_INPUT(z);
  CHECK_CUDA_INPUT(dz);
  CHECK_CUDA_INPUT(weight);
  CHECK_CUDA_INPUT(bias);

  // Extract dimensions
  int64_t num, chn, sp;
  get_dims(z, num, chn, sp);

  auto edz = at::empty({chn},z.options().dtype(at::kFloat));
  auto eydz = at::empty({chn},z.options().dtype(at::kFloat));

  // Run kernel
  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  auto stream = at::cuda::getCurrentCUDAStream();
  edz_eydz_kernel_h<<<blocks, threads, 0, stream>>>(
        z.data_ptr<at::Half>(),
        dz.data_ptr<at::Half>(),
        weight.data_ptr<float>(),
        bias.data_ptr<float>(),
        edz.data_ptr<float>(),
        eydz.data_ptr<float>(),
        affine, eps, num, chn, sp);
 
  return {edz, eydz};
}

__global__ void backward_kernel_h(const c10::Half *z, const c10::Half *dz, const float *var, const float *weight, const float *bias, 
                                const float *edz, const float *eydz, c10::Half *dx, bool affine, float eps, int num, int chn, int sp) {
  int plane = blockIdx.x;

  float _weight = affine ? abs(weight[plane]) + eps : 1.f;
  float _bias = affine ? bias[plane] : 0.f;
  float _var = var[plane];
  float _edz = edz[plane];
  float _eydz = eydz[plane];

  float _mul = _weight * rsqrt(_var + eps);
  float count = float(num * sp);

  for (int batch = 0; batch < num; ++batch) {
    for (int n = threadIdx.x; n < sp; n += blockDim.x) {
      float _dz = __half2float(*reinterpret_cast<const half*>(&dz[(batch * chn + plane) * sp + n]));
      float _y = (__half2float(*reinterpret_cast<const half*>(&z[(batch * chn + plane) * sp + n])) - _bias) / _weight;

      *reinterpret_cast<half*>(&dx[(batch * chn + plane) * sp + n]) = __float2half((_dz - _edz / count - _y * _eydz / count) * _mul);
    }
  }
}

at::Tensor backward_cuda_h(at::Tensor z, at::Tensor dz, at::Tensor var, at::Tensor weight, at::Tensor bias,
                                      at::Tensor edz, at::Tensor eydz, bool affine, float eps) {
  CHECK_CUDA_INPUT(z);
  CHECK_CUDA_INPUT(dz);
  CHECK_CUDA_INPUT(var);
  CHECK_CUDA_INPUT(weight);
  CHECK_CUDA_INPUT(bias);
  CHECK_CUDA_INPUT(edz);
  CHECK_CUDA_INPUT(eydz);

  // Extract dimensions
  int64_t num, chn, sp;
  get_dims(z, num, chn, sp);

  auto dx = at::zeros_like(z);

  // Run kernel
  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  auto stream = at::cuda::getCurrentCUDAStream();
  backward_kernel_h<<<blocks, threads, 0, stream>>>(
        z.data_ptr<at::Half>(),
        dz.data_ptr<at::Half>(),
        var.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias.data_ptr<float>(),
        edz.data_ptr<float>(),
        eydz.data_ptr<float>(),
        dx.data_ptr<at::Half>(),
        affine, eps, num, chn, sp);

  return dx;
}

__global__ void leaky_relu_backward_impl_h(c10::Half *z, c10::Half *dz, float slope, int64_t count) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count;  i += blockDim.x * gridDim.x){
    float _z = __half2float(*reinterpret_cast<half*>(&z[i]));
    if (_z < 0) {
      *reinterpret_cast<half*>(&dz[i]) = __float2half(__half2float(*reinterpret_cast<half*>(&dz[i])) * slope);
      *reinterpret_cast<half*>(&z[i]) = __float2half(_z / slope);
    }
  }
}

void leaky_relu_backward_cuda_h(at::Tensor z, at::Tensor dz, float slope) {
  CHECK_CUDA_INPUT(z);
  CHECK_CUDA_INPUT(dz);

  int64_t count = z.numel();
  dim3 threads(getNumThreads(count));
  dim3 blocks = (count + threads.x - 1) / threads.x;
  auto stream = at::cuda::getCurrentCUDAStream();
  leaky_relu_backward_impl_h<<<blocks, threads, 0, stream>>>(
      z.data_ptr<at::Half>(),
      dz.data_ptr<at::Half>(),
      slope, count);
}

